#include "hip/hip_runtime.h"
/*
 * @Author: xpt 
 * @Date: 2019-12-17 22:43:29 
 * @Last Modified by: xpt
 * @Last Modified time: 2019-12-20 17:44:36
 */
 #include<iostream>
 #include<string>
 #include<stdlib.h>
 #include"../Tools/Macro.h"
 #include"../Tools/Timeit.h"
 #include"../cudaLib/cudaCommon.h"
 #include"../Tools/PdwStruct/PdwStruct.hpp"
 using namespace std;

#define UNROLL_NUMS 8

__constant__ int d_ArraySize, d_SharedSize, d_Order;

/* @Author: xpt
 * 1.__launch_bounds__(maxThreadsPerBlock, minBlockPerMultiprocessor) 
 * maxThreadsPerBlock指出了每个线程块可以包含的最大线程数，这个线程块由核函数来启动。
 * minBlockPerMultiprocessor是可选参数，指明了在每个SM中预期的最小的常驻线程块数量。
 * 2.__restrict__  只读缓存，可以使用内部函数__ldg用于代替标准指针解引用。
 * 3.共享内存和一级缓存共用，可以使用hipDeviceSetCacheConfig配置大小。
 * 4.使用常量索引的局部数组变量，也可能分配在寄存器上，在编译时使用 -Xptxas -v, -abi=no
 * 可以输出寄存器的数量、共享内存的字节数以及每个线程所使用的常量内存的字节数。
 * 你还可以使用maxrregcount编译器选项(-maxrregcount=32)，来控制一个编译单元里所有核
 * 函数使用的寄存器的最大数量。
 * 5.#pragma unroll 编译时循环展开。
 * 6.还使用了防止线程束分化、显式障碍、内存栅栏、弱排序、内存填充、指令洗牌等技术。
 */
template<class Ty, int UnrollNums>
__global__ void
__launch_bounds__(256,2)
HistogramDiff(const Ty * __restrict__ inArray, Ty *OutArray)
{
    extern __shared__ Ty Diff[];                                                    //一阶导(x2-x1)/dx SharedSize+冗余Order 使用共享内存 建议？禁用一级Cache -dlcm=cg
    Ty DiffTmp[UnrollNums];                                                         //这个操作或许？可以使用寄存器，64个？256个？没有使用寄存器，大概率是在二级Cache上了。
    UINT InArrIdx           = UnrollNums*(blockIdx.x * blockDim.x + threadIdx.x);   //当前线程处理输入数组起始索引值
    UINT SmemIdx            = InArrIdx % d_SharedSize;                              //当前线程负责计算一级阶差的共享内存偏移
    UINT rearOrderThreadsNo = d_SharedSize - d_Order;                               //冗余Order数的一级阶差由Smem偏移线程打理
    #pragma unroll
    for(int i = 0; i < UnrollNums; i++)
    {
        DiffTmp[i]      = inArray[InArrIdx+i+1] - inArray[InArrIdx+i];
        Diff[SmemIdx+i] = DiffTmp[i];
        OutArray[InArrIdx+i] = DiffTmp[i];
    }
    if(threadIdx.x >= rearOrderThreadsNo/UnrollNums)                                //线程束不会分化，然而断言判断不会产生？尝试 -g 和 O3, $结果写回
    {
        #pragma unroll
        for(int i = 0; i < UnrollNums; i++)
        {
            Diff[SmemIdx+d_Order+i] = inArray[InArrIdx+d_Order+i+1] - inArray[InArrIdx+d_Order+i];
        }
    }
    __syncthreads();
    for(int i = 1; i < d_Order; i++)
    {
        #pragma unroll
        for(int j = 0; j < UnrollNums; j++)
        {
            DiffTmp[j] = DiffTmp[j] + Diff[SmemIdx+j+i];
            OutArray[i*d_ArraySize+InArrIdx+j] = DiffTmp[j];
        }
    }
}

template<class Ty>
void HistogramDiffOnHost(const Ty * inArray, Ty *OutArray, int ArraySize, int Difflens, int Order)
{
    for(int i = 0; i < Order; i++)
    {
        for(int j = 0; j < Difflens-Order-1; j++)
        {
            OutArray[i*ArraySize + j] = inArray[j+i+1] - inArray[j];
        }
    }
}

template<class Ty>
void SameCheckHistogram(const Ty * inArray0, const Ty * inArray1, const Ty * inArray2, int ArraySize, int Difflens, int Order)
{
    bool CheckTag = true;
    for(int i = 0; i < Order; i++)
    {
        for(int j = 0; j < Difflens-Order-1; j++)
        {
            if(inArray1[i*ArraySize + j] != inArray2[i*ArraySize + j])
            {
                COUT(i,j);
                inArray0[j].print();
                inArray0[j+i+1].print();
                inArray1[i*ArraySize + j].print();
                inArray2[i*ArraySize + j].print();
                CheckTag = false;
                break;
            }
        }
        if(CheckTag == false)
        {
            break;
        }
    }
}

template<class Ty>
void cHistogramDiff(int GridSize, int BlockSize, const Ty *d_pInArray, Ty *d_pOutArray, int SharedMemSize, int Order)
{
    COUTS("SharedMemSize: ", (SharedMemSize+Order) * sizeof(Ty));
    HistogramDiff<Ty, UNROLL_NUMS><<<GridSize, BlockSize, (SharedMemSize+Order) * sizeof(Ty)>>>(d_pInArray, d_pOutArray);
}

typedef _Pdw<> _PdwTy;

int main(int argc, char **argv)
{
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    COUTS("starting reduction at ",argv[0]);
    COUTS("device no: ", dev, "device name: ", deviceProp.name);
    CHECK(hipSetDevice(dev));
    CHECK(hipDeviceSetCacheConfig(hipFuncCachePreferShared));

    int Order       = 128;
    int blocksize   = 256;
    int SharedMems  = 1024;
    int size        = 1<<19;
    switch (argc)
    {
    case 4:
        size        = atoi(argv[3]);
    case 3:
        SharedMems  = atoi(argv[2]);
    case 2:
        Order       = atoi(argv[1]);
    case 1:
    default:
        blocksize = SharedMems/UNROLL_NUMS;
        COUT(Order, blocksize, SharedMems, size);
        break;
    }
    setupConstant(d_ArraySize, size);
    setupConstant(d_SharedSize, SharedMems);
    setupConstant(d_Order, Order);

    dim3 block (blocksize, 1);
    dim3 grid  ((size + block.x - 1) / block.x, 1);
    COUTS("grid: ", grid.x, "block: ", block.x);

    UINT InArrPdwbytes  = size * sizeof(_PdwTy);
    UINT OutArrPdwbytes = Order * InArrPdwbytes;

    _PdwTy *pInPdwArray;
    _PdwTy *pOutPdwArray;

    _Timeit Timeit(6);
    Timeit.Start(0, "Host hipHostMalloc allocation took(us) ");
    CHECK(hipHostMalloc ((_PdwTy **)&pInPdwArray, InArrPdwbytes));
    CHECK(hipHostMalloc ((_PdwTy **)&pOutPdwArray, OutArrPdwbytes));
    Timeit.End(0);

    for(int i = 0; i < size; i++)
    {
        pInPdwArray[i].random();
    }

    _PdwTy *d_pInPdwArray = NULL;
    _PdwTy *d_pOutPdwArray = NULL;

    CHECK(hipMalloc((void **) &d_pInPdwArray, InArrPdwbytes));
    CHECK(hipMalloc((void **) &d_pOutPdwArray, OutArrPdwbytes));

    CHECK(hipMemcpy(d_pInPdwArray, pInPdwArray, InArrPdwbytes, hipMemcpyHostToDevice));

    Timeit.Start(1, "HistogramDiff took(us) ");
    cHistogramDiff(grid.x/UNROLL_NUMS - 1, block.x, d_pInPdwArray, d_pOutPdwArray, SharedMems, Order);
    CHECK(hipDeviceSynchronize());
    Timeit.End(1);
    CHECK(hipMemcpy(pOutPdwArray, d_pOutPdwArray, OutArrPdwbytes, hipMemcpyDeviceToHost));

    _PdwTy *pHostOutPdwArray = (_PdwTy *)malloc(OutArrPdwbytes);
    Timeit.Start(2, "HistogramDiffOnHost took(us) ");
    HistogramDiffOnHost(pInPdwArray, pHostOutPdwArray, size, UNROLL_NUMS * (grid.x/UNROLL_NUMS-1) * block.x, Order);
    Timeit.End(2);
    Timeit.Show(0,6);
    SameCheckHistogram(pInPdwArray, pOutPdwArray, pHostOutPdwArray, size, UNROLL_NUMS * (grid.x/UNROLL_NUMS-1) * block.x, Order);

    free(pHostOutPdwArray);
    CHECK(hipHostFree(pInPdwArray));
    CHECK(hipHostFree(pOutPdwArray));
    CHECK(hipFree(d_pInPdwArray));
    CHECK(hipFree(d_pOutPdwArray));
    hipDeviceReset();
    return EXIT_SUCCESS;
}