#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>

#include "helper_functions.h"
#include <hip/hip_runtime_api.h>

#include "rsp_base.cuh"

// Complex addition
static __device__ __host__ inline Complex ComplexAdd(Complex a, Complex b)
{
	Complex c;
	c.x = a.x + b.x;
	c.y = a.y + b.y;
	return c;
}

// Complex scale
static __device__ __host__ inline Complex ComplexScale(Complex a, float s)
{
	Complex c;
	c.x = s * a.x;
	c.y = s * a.y;
	return c;
}

// Complex multiplication
static __device__ __host__ inline Complex ComplexMul(Complex a, Complex b)
{
	Complex c;
	c.x = a.x * b.x - a.y * b.y;
	c.y = a.x * b.y + a.y * b.x;
	return c;
}

// Complex multiplication
static __device__ __host__ inline Complex ComplexMulReal(Complex a, float b)
{
	Complex c;
	c.x = a.x * b;
	c.y = a.y * b;
	return c;
}


// Complex multiplication
static __device__ __host__ inline float ComplexAbs(Complex a)
{
	float c;

	c = fabsf(a.x * a.x + a.y * a.y);

	return c;
}


// Complex pointwise multiplication
__global__ void ComplexPointwiseMulAndScale(Complex *a, const Complex *b, int size, float scale)
{
	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i = threadID; i < size; i += numThreads)
	{
		a[i] = ComplexScale(ComplexMul(a[i], b[i]), scale);
	}
}



//static __global__ void MatrixWiseMulVec(Complex* a, const Complex *b, int row, int col, float scale)
//{
//
//}

__global__ void MatrixWiseMulVecRow(Complex* m, Complex *v, Complex* result, int row, int col, float scale)
{
	int crow, ccol;
	//int cstride;
	int offset;
	Complex vv;

	crow = threadIdx.x;
	ccol = blockIdx.x;
	offset = crow*col + ccol;

	crow = blockIdx.y * blockDim.y + threadIdx.y;
	ccol = blockIdx.x * blockDim.x + threadIdx.x;
	//cstride = blockDim.x * gridDim.x;
	offset = crow * col + ccol;

	if ((crow < row) && (ccol < col))
	{
		vv = ComplexMulReal(v[ccol], scale);
		result[offset] = ComplexMul(m[offset], vv);
	}
}

__global__ void MatrixWiseRealMulVecCol(Complex* m, float *v, Complex* result, int row, int col)
{
	int crow, ccol;
	//int cstride;
	int offset;

	crow = threadIdx.x;
	ccol = blockIdx.x;
	offset = crow*col + ccol;

	crow = blockIdx.y * blockDim.y + threadIdx.y;
	ccol = blockIdx.x * blockDim.x + threadIdx.x;
	//cstride = blockDim.x * gridDim.x;
	offset = crow * col + ccol;

	if ((crow < row) && (ccol < col))
	{
		result[offset] = ComplexMulReal(m[offset], v[crow]);
	}
}


void rsp_FFTRow(hipfftComplex *fft_datain, hipfftComplex *fft_dataout, int fft_row, int fft_col)
{

	// CUFFT plan advanced API
	hipfftHandle plan_adv;
	size_t workSize;
	int fft_size = fft_col;
	int new_size_long = fft_size;
	int batch_size = fft_row;

	int nembed[2] = { fft_row , fft_col };
	int stride = 1;
	int dist = fft_col;

	checkCudaErrors(hipfftCreate(&plan_adv));
	checkCudaErrors(hipfftMakePlanMany(plan_adv, 1, &new_size_long, nembed, stride, dist, nembed, stride, dist, HIPFFT_C2C, batch_size, &workSize));

	// Transform signal and kernel
	checkCudaErrors(hipfftExecC2C(plan_adv, (hipfftComplex *)fft_datain, (hipfftComplex *)fft_dataout, HIPFFT_FORWARD));

	//Destroy CUFFT context
	checkCudaErrors(hipfftDestroy(plan_adv));



}

void rsp_FFTCol(hipfftComplex *fft_datain, hipfftComplex *fft_dataout, int fft_row, int fft_col)
{

	int fft_size = fft_row;

	// CUFFT plan advanced API
	hipfftHandle plan_adv;
	size_t workSize;
	int new_size_long = fft_size;
	int batch_size = fft_col;

	int nembed[2] = { fft_row , fft_col };
	int stride = fft_col;
	int dist = 1;


	checkCudaErrors(hipfftCreate(&plan_adv));
	checkCudaErrors(hipfftMakePlanMany(plan_adv, 1, &new_size_long, nembed, stride, dist, nembed, stride, dist, HIPFFT_C2C, batch_size, &workSize));

	// Transform signal and kernel
	checkCudaErrors(hipfftExecC2C(plan_adv, (hipfftComplex *)fft_datain, (hipfftComplex *)fft_dataout, HIPFFT_FORWARD));

	//Destroy CUFFT context
	checkCudaErrors(hipfftDestroy(plan_adv));
}