#include <stdio.h>
#include <hip/hip_runtime.h>
 
 int main(void)
 {
     hipError_t err = hipSuccess;
 
     //* cudaMalloc3D
     hipPitchedPtr  cudaPitchedPtrCase;
     hipExtent cudaExtentCase = {900, 800, 513};
     err = hipMalloc3D(&cudaPitchedPtrCase,cudaExtentCase);
     if (err != hipSuccess)
     {
         fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
         exit(EXIT_FAILURE);
     }
     printf("cudaPitchedPtrCase.pitch = %d\n",cudaPitchedPtrCase.pitch);
     printf("cudaPitchedPtrCase.xsize = %d\n",cudaPitchedPtrCase.xsize);
     printf("cudaPitchedPtrCase.ysize = %d\n",cudaPitchedPtrCase.ysize);
     printf("Done\n");

    //* cudaMallocPitch
    // void *cudaptr;
    // size_t pitch;
    // cudaMallocPitch((float **)&cudaptr,&pitch,100,900);
    // printf("pitch = %d\n",pitch);
    // cudaFree(cudaptr);
     return 0;
 }
 
 