#include "hip/hip_runtime.h"
/*
* Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
*
* Please refer to the NVIDIA end user license agreement (EULA) associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms of the EULA
* is strictly prohibited.
*
*/

/*
* Copyright 1993-2014 NVIDIA Corporation.  All rights reserved.
*
* Please refer to the NVIDIA end user license agreement (EULA) associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms of the EULA
* is strictly prohibited.
*
*/

/* Example showing the use of CUFFT for fast 1D-convolution using FFT. */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>

#include "helper_functions.h"
#include <hip/hip_runtime_api.h>

#include "rsp_base.cuh"
#include "radar_process.h"






////////////////////////////////////////////////////////////////////////////////

// The filter size is assumed to be a number smaller than the signal size
#define SIGNAL_ROW	        16
#define SIGNAL_COL			64

////////////////////////////////////////////////////////////////////////////////
// Complex operations
////////////////////////////////////////////////////////////////////////////////

// Complex addition
//static __device__ __host__ inline Complex ComplexAdd(Complex a, Complex b)
//{
//	Complex c;
//	c.x = a.x + b.x;
//	c.y = a.y + b.y;
//	return c;
//}
//
//// Complex scale
//static __device__ __host__ inline Complex ComplexScale(Complex a, float s)
//{
//	Complex c;
//	c.x = s * a.x;
//	c.y = s * a.y;
//	return c;
//}
//
//// Complex multiplication
//static __device__ __host__ inline Complex ComplexMul(Complex a, Complex b)
//{
//	Complex c;
//	c.x = a.x * b.x - a.y * b.y;
//	c.y = a.x * b.y + a.y * b.x;
//	return c;
//}
//
//// Complex multiplication
//static __device__ __host__ inline Complex ComplexMulReal(Complex a, float b)
//{
//	Complex c;
//	c.x = a.x * b;
//	c.y = a.y * b;
//	return c;
//}
//
//
//// Complex multiplication
//static __device__ __host__ inline float ComplexAbs(Complex a)
//{
//	float c;
//	
//	c = fabsf(a.x * a.x + a.y * a.y);
//
//	return c;
//}
//
//
//// Complex pointwise multiplication
//static __global__ void ComplexPointwiseMulAndScale(Complex *a, const Complex *b, int size, float scale)
//{
//	const int numThreads = blockDim.x * gridDim.x;
//	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
//
//	for (int i = threadID; i < size; i += numThreads)
//	{
//		a[i] = ComplexScale(ComplexMul(a[i], b[i]), scale);
//	}
//}
//
//
//
////static __global__ void MatrixWiseMulVec(Complex* a, const Complex *b, int row, int col, float scale)
////{
////
////}
//
//static __global__ void MatrixWiseMulVecRow(Complex* m, Complex *v, Complex* result, int row, int col, float scale)
//{
//	int crow, ccol;
//	//int cstride;
//	int offset;
//	Complex vv;
//
//	crow = threadIdx.x;
//	ccol = blockIdx.x;
//	offset = crow*col + ccol;
//
//	crow = blockIdx.y * blockDim.y + threadIdx.y;
//	ccol = blockIdx.x * blockDim.x + threadIdx.x;
//	//cstride = blockDim.x * gridDim.x;
//	offset = crow * col + ccol;
//
//	if ((crow < row) && (ccol < col))
//	{
//		vv = ComplexMulReal(v[ccol],scale);
//		result[offset] = ComplexMul(m[offset], vv);
//	}
//}
//
//static __global__ void MatrixWiseRealMulVecCol(Complex* m, float *v, Complex* result, int row, int col)
//{
//	int crow, ccol;
//	//int cstride;
//	int offset;
//
//	crow = threadIdx.x;
//	ccol = blockIdx.x;
//	offset = crow*col + ccol;
//
//	crow = blockIdx.y * blockDim.y + threadIdx.y;
//	ccol = blockIdx.x * blockDim.x + threadIdx.x;
//	//cstride = blockDim.x * gridDim.x;
//	offset = crow * col + ccol;
//
//	if ((crow < row) && (ccol < col))
//	{
//		result[offset] = ComplexMulReal(m[offset], v[crow]);
//	}
//}


void rsp_blackman(int nSize, float* pWin)
{
	int i;

	for (i = 0; i < nSize; i++)
	{
		pWin[i] = 0.52 - 0.5*cos(2 * pi*i / (nSize - 1)) + 0.08*cos(4*pi*i/(nSize-1));
	}
}

void pulse_compress(float* rx, const float* win, int row, int col)
{
	
	int M_ROW = 32;
	int M_COL = 8192;

	Complex* fp_rx, *fp_result;
	float* fp_win;
	Complex* dev_rx, *dev_result;
	float* dev_win;

	hipEvent_t start;
	hipEvent_t stop;
	hipStream_t stream0;
	float ElapsedTime;

	const int threadsPerBlock = M_ROW;
	const int blocksPerGrid = M_COL;


	int count;
	hipDeviceProp_t prop;

	checkCudaErrors(hipGetDeviceCount(&count));

	checkCudaErrors(hipGetDeviceProperties(&prop, 0));

	fp_rx = (Complex*)malloc(M_ROW * M_COL * sizeof(Complex));
	fp_win = (float*)malloc(M_COL * sizeof(float));
	fp_result = (Complex*)malloc(M_ROW * M_COL * sizeof(Complex));

	checkCudaErrors(hipMalloc((void**)&dev_rx, M_ROW * M_COL * sizeof(Complex)));
	checkCudaErrors(hipMalloc((void**)&dev_win, M_ROW * M_COL * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&dev_result, M_ROW * M_COL * sizeof(Complex)));

	for (int i = 0; i < M_ROW; i++)
	{
		for (int j = 0; j < M_COL; j++)
		{
			fp_rx[i*M_COL + j].x = (float)j;
			fp_rx[i*M_COL + j].y = 0;
		}

	}

	for (int j = 0; j < M_COL; j++)
		fp_win[j] = (float)j;


	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));
	checkCudaErrors(hipEventRecord(start, 0));


	checkCudaErrors(hipMemcpy(dev_rx, fp_rx, M_ROW * M_COL * sizeof(Complex), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dev_win, fp_win, M_COL * sizeof(float), hipMemcpyHostToDevice));
	
	//MatrixWiseMulVec <<<blocksPerGrid, threadsPerBlock >>>(dev_rx, dev_win, dev_result, M_ROW, M_COL, 1.0f);

	checkCudaErrors(hipMemcpy(fp_result, dev_result, M_ROW * M_COL * sizeof(Complex), hipMemcpyDeviceToHost));

	checkCudaErrors(hipEventRecord(stop, 0));
	checkCudaErrors(hipEventSynchronize(stop));

	checkCudaErrors(hipEventElapsedTime(&ElapsedTime, start, stop));





	checkCudaErrors(hipFree(dev_result));
	checkCudaErrors(hipFree(dev_win));
	checkCudaErrors(hipFree(dev_rx));

	free(fp_result);
	free(fp_win);
	free(fp_rx);


		

}


void FFTTest(int fft_row, int fft_col�� float* run_time)
{
	hipEvent_t start;
	hipEvent_t stop;
	hipStream_t stream0;
	float ElapsedTime;

	LARGE_INTEGER lstart, lstop, ltotal;

	int buffer_size = fft_row * fft_col;
	int fft_size = fft_col;
	// Allocate host memory for the signal
	Complex *h_signal = (Complex *)malloc(sizeof(Complex) * buffer_size);
	Complex *h_fft = (Complex *)malloc(sizeof(Complex) * buffer_size);

	// Initialize the memory for the signal
	for (unsigned int i = 0; i < fft_row; ++i)
		for (unsigned int j = 0; j < fft_col; ++j)
	{
		h_signal[i*fft_col + j].x = (float)i+1.0f;
		h_signal[i*fft_col + j].y = 0;
	}



	int mem_size = sizeof(Complex) * buffer_size;

	// Allocate device memory for signal
	Complex *d_signal;
	checkCudaErrors(hipMalloc((void **)&d_signal, mem_size));



	// CUFFT plan advanced API
	hipfftHandle plan_adv;
	size_t workSize;
	int new_size_long = fft_size;
	int batch_size = fft_row;

	int nembed[2] = { fft_row , fft_col };
	int stride = 1;
	int dist = fft_col;

	QueryPerformanceCounter( &lstart);

	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));
	checkCudaErrors(hipEventRecord(start, 0));

	// Copy host memory to device
	checkCudaErrors(hipMemcpy(d_signal, h_signal, mem_size,
		hipMemcpyHostToDevice));

	checkCudaErrors(hipfftCreate(&plan_adv));
	checkCudaErrors(hipfftMakePlanMany(plan_adv, 1, &new_size_long, nembed, stride, dist, nembed, stride, dist, HIPFFT_C2C, batch_size, &workSize));

	// Transform signal and kernel
	checkCudaErrors(hipfftExecC2C(plan_adv, (hipfftComplex *)d_signal, (hipfftComplex *)d_signal, HIPFFT_FORWARD));

	// Copy device memory to host
	checkCudaErrors(hipMemcpy(h_fft, d_signal, mem_size,
		hipMemcpyDeviceToHost));

	checkCudaErrors(hipEventRecord(stop, 0));
	checkCudaErrors(hipEventSynchronize(stop));

	QueryPerformanceCounter(&lstop);
	checkCudaErrors(hipEventElapsedTime(&ElapsedTime, start, stop));

	ltotal.QuadPart = (lstop.QuadPart - lstart.QuadPart) / batch_size;
	ElapsedTime = ElapsedTime / batch_size;

	*run_time = ElapsedTime;
	// Transform signal back
	//checkCudaErrors(hipfftExecC2C(plan_adv, (hipfftComplex *)d_signal, (hipfftComplex *)d_signal, HIPFFT_BACKWARD));

	// Copy device memory to host
	checkCudaErrors(hipMemcpy(h_signal, d_signal, mem_size,
		hipMemcpyDeviceToHost));


	//Destroy CUFFT context
	checkCudaErrors(hipfftDestroy(plan_adv));

	// cleanup memory
	free(h_signal);
	free(h_fft);
	checkCudaErrors(hipFree(d_signal));

	
}

void ColFFTTest(int fft_row, int fft_col)
{
	hipEvent_t start;
	hipEvent_t stop;
	hipStream_t stream0;
	float ElapsedTime;

	LARGE_INTEGER lstart, lstop, ltotal;
	Complex *h_signal;
	Complex *h_fft;
	Complex *d_signal;
	Complex *d_fft;

	int buffer_size = fft_row * fft_col;
	int fft_size = fft_row;
	// Allocate host memory for the signal
	//Complex *h_signal = (Complex *)malloc(sizeof(Complex) * buffer_size);
	//Complex *h_fft = (Complex *)malloc(sizeof(Complex) * buffer_size);

	checkCudaErrors(hipHostAlloc((void**)&h_signal, sizeof(Complex) * buffer_size, hipHostMallocDefault));
	checkCudaErrors(hipHostAlloc((void**)&h_fft, sizeof(Complex) * buffer_size, hipHostMallocDefault));


	// Initialize the memory for the signal
	for (unsigned int i = 0; i < fft_row; ++i)
		for (unsigned int j = 0; j < fft_col; ++j)
		{
			h_signal[i*fft_col + j].x = (float)j + 1.0f;
			h_signal[i*fft_col + j].y = 0;
		}



	int mem_size = sizeof(Complex) * buffer_size;

	// Allocate device memory for signal
	
	checkCudaErrors(hipMalloc((void **)&d_signal, mem_size));
	checkCudaErrors(hipMalloc((void **)&d_fft, mem_size));



	// CUFFT plan advanced API
	hipfftHandle plan_adv;
	size_t workSize;
	int new_size_long = fft_size;
	int batch_size = fft_col;

	int nembed[2] = { fft_row , fft_col };
	int stride = fft_col;
	int dist = 1;

	checkCudaErrors(hipStreamCreate(&stream0));


	QueryPerformanceCounter(&lstart);

	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));
	checkCudaErrors(hipEventRecord(start, 0));

	// Copy host memory to device
	//checkCudaErrors(hipMemcpy(d_signal, h_signal, mem_size, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpyAsync(d_signal, h_signal, mem_size, hipMemcpyHostToDevice, stream0));

	checkCudaErrors(hipfftCreate(&plan_adv));
	checkCudaErrors(hipfftMakePlanMany(plan_adv, 1, &new_size_long, nembed, stride, dist, nembed, stride, dist, HIPFFT_C2C, batch_size, &workSize));

	// Transform signal and kernel
	checkCudaErrors(hipfftExecC2C(plan_adv, (hipfftComplex *)d_signal, (hipfftComplex *)d_fft, HIPFFT_FORWARD));

	// Copy device memory to host
	checkCudaErrors(hipMemcpyAsync(h_fft, d_fft, mem_size, hipMemcpyDeviceToHost, stream0));

	// Transform signal back
	checkCudaErrors(hipfftExecC2C(plan_adv, (hipfftComplex *)d_fft, (hipfftComplex *)d_signal, HIPFFT_BACKWARD));

	// Copy device memory to host
	checkCudaErrors(hipMemcpyAsync(h_signal, d_signal, mem_size, hipMemcpyDeviceToHost, stream0));
	checkCudaErrors(hipStreamSynchronize(stream0));

	// Stop Time
	//checkCudaErrors(hipStreamSynchronize(stream0));

	QueryPerformanceCounter(&lstop);
	checkCudaErrors(hipEventRecord(stop, 0));
	checkCudaErrors(hipEventSynchronize(stop));

	ltotal.QuadPart = (lstop.QuadPart - lstart.QuadPart) / batch_size;

	checkCudaErrors(hipEventElapsedTime(&ElapsedTime, start, stop));

	ElapsedTime = ElapsedTime / batch_size;

	//Destroy CUFFT context
	checkCudaErrors(hipfftDestroy(plan_adv));

	checkCudaErrors(hipStreamDestroy(stream0));

	// cleanup memory
	checkCudaErrors(hipHostFree(h_signal));
	checkCudaErrors(hipHostFree(h_fft));
	checkCudaErrors(hipFree(d_signal));
	checkCudaErrors(hipFree(d_fft));


}

void rsp_Fft1d(Complex* pSrc, Complex* pDst, int nFftSize)
{
	hipfftHandle plan;
	Complex* d_fftbuf;

	checkCudaErrors(hipMalloc((void **)&d_fftbuf, nFftSize));
	checkCudaErrors(hipfftPlan1d(&plan, nFftSize, HIPFFT_C2C, 1));

	checkCudaErrors(hipMemcpy(d_fftbuf, pSrc, nFftSize, hipMemcpyHostToDevice));

	checkCudaErrors(hipfftExecC2C(plan, (hipfftComplex *)d_fftbuf, (hipfftComplex *)d_fftbuf, HIPFFT_FORWARD));

	checkCudaErrors(hipMemcpy(pDst, d_fftbuf, nFftSize, hipMemcpyDeviceToHost));

	checkCudaErrors(hipfftDestroy(plan));
	checkCudaErrors(hipFree(d_fftbuf));
}

void rsp_MemcpyToDevice(Complex* pDevBuf, void* pHost, int size, hipStream_t stream)
{
	checkCudaErrors(hipMemcpyAsync(pDevBuf, pHost, size, hipMemcpyHostToDevice, stream));
}

void rsp_MemcpyToHost(void* pHost, Complex* pDevBuf, int size, hipStream_t stream)
{
	checkCudaErrors(hipMemcpyAsync(pHost, pDevBuf, size, hipMemcpyDeviceToHost, stream));
}

void rsp_MatchFilter_Generator(RSP_CUDE_ENV* pCudaEnv, int nIdx,  int nCode, float fBw, float fPw, float fFs, UINT32 uOffset, int nLen)
{
	pCudaEnv->pMatchFilter[nIdx].nCode = nCode;
	pCudaEnv->pMatchFilter[nIdx].fBw = fBw;
	pCudaEnv->pMatchFilter[nIdx].fPw = fPw;
	pCudaEnv->pMatchFilter[nIdx].fFs = fFs;
	pCudaEnv->pMatchFilter[nIdx].uOffset = uOffset;
	pCudaEnv->pMatchFilter[nIdx].nLen = nLen;

	Complex* pBuf = (Complex*)(&pCudaEnv->pHostMatchFilter[uOffset]);

	int nPw = floor(fPw * fFs);
	float k = fBw / fPw;
	float ts = 1 / fFs;

	for (int i = nPw - 1; i >= 0; i--)
	{
		float t = -fPw / 2 + i * ts;
		pBuf[i].x = cos(-pi*k*t*t);
		pBuf[i].y = sin(-pi*k*t*t);
	}

	rsp_Fft1d(pBuf, pBuf, nLen);
}

void rsp_MatchFilter_Init(RSP_CUDE_ENV* pCudaEnv)
{
	pCudaEnv->MF_NUM = CODE_NUM;
	pCudaEnv->pMatchFilter = (RSP_MATCHFILTER*)malloc(pCudaEnv->MF_NUM * sizeof(RSP_MATCHFILTER));

	rsp_MatchFilter_Generator(pCudaEnv, 0, 1, 5e6, 6.4e-6, 10e6, 0, 65536);
	rsp_MatchFilter_Generator(pCudaEnv, 1, 1, 5e6, 6.4e-6, 10e6, pCudaEnv->pMatchFilter[0].uOffset + pCudaEnv->pMatchFilter[0].nLen, 32768);
	rsp_MatchFilter_Generator(pCudaEnv, 2, 1, 5e6, 6.4e-6, 10e6, pCudaEnv->pMatchFilter[1].uOffset + pCudaEnv->pMatchFilter[1].nLen, 16384);
	rsp_MatchFilter_Generator(pCudaEnv, 3, 1, 5e6, 6.4e-6, 10e6, pCudaEnv->pMatchFilter[2].uOffset + pCudaEnv->pMatchFilter[2].nLen, 8192);
	rsp_MatchFilter_Generator(pCudaEnv, 4, 1, 5e6, 6.4e-6, 10e6, pCudaEnv->pMatchFilter[3].uOffset + pCudaEnv->pMatchFilter[3].nLen, 4096);
	rsp_MatchFilter_Generator(pCudaEnv, 5, 1, 5e6, 6.4e-6, 10e6, pCudaEnv->pMatchFilter[4].uOffset + pCudaEnv->pMatchFilter[4].nLen, 2048);
	rsp_MatchFilter_Generator(pCudaEnv, 6, 1, 5e6, 6.4e-6, 10e6, pCudaEnv->pMatchFilter[5].uOffset + pCudaEnv->pMatchFilter[5].nLen, 1024);
	rsp_MatchFilter_Generator(pCudaEnv, 7, 1, 5e6, 6.4e-6, 10e6, pCudaEnv->pMatchFilter[6].uOffset + pCudaEnv->pMatchFilter[6].nLen, 512);
	rsp_MatchFilter_Generator(pCudaEnv, 8, 1, 5e6, 6.4e-6, 10e6, pCudaEnv->pMatchFilter[7].uOffset + pCudaEnv->pMatchFilter[7].nLen, 256);

	rsp_MemcpyToDevice(pCudaEnv->pDevMatchFilter, pCudaEnv->pHostMatchFilter, pCudaEnv->MF_NUM * sizeof(RSP_MATCHFILTER), pCudaEnv->stream0);
}


void rsp_Win_Init(RSP_CUDE_ENV* pCudaEnv)
{
	float* pHostWin;

	int total_size = 0;
	for (total_size = 16; total_size <= 65536; total_size *= 2);

	pHostWin = (float*)malloc(sizeof(float)*total_size);

	for (int win_size = 16; win_size <= 65536; win_size *= 2)
	{
		rsp_blackman(win_size, &pHostWin[total_size]);
		total_size += win_size;
	}

	checkCudaErrors(hipMemcpy(pCudaEnv->pBlackmanWin, pHostWin, total_size, hipMemcpyHostToDevice));

	free(pHostWin);
}
	

void rsp_Init(RSP_CUDE_ENV* pCudaEnv)
{
	checkCudaErrors(hipHostAlloc((void**)&pCudaEnv->pHostMatchFilter, sizeof(Complex) * MATCHFILTER_BUFFER_SIZE, hipHostMallocDefault));
	checkCudaErrors(hipHostAlloc((void**)&pCudaEnv->pHostRxData, sizeof(Complex) * DATA_BUFFER_SIZE, hipHostMallocDefault));
	checkCudaErrors(hipHostAlloc((void**)&pCudaEnv->pHostPcData, sizeof(Complex) * DATA_BUFFER_SIZE, hipHostMallocDefault));
	checkCudaErrors(hipHostAlloc((void**)&pCudaEnv->pHostMTDData, sizeof(Complex) * DATA_BUFFER_SIZE, hipHostMallocDefault));

	checkCudaErrors(hipMalloc((void**)&pCudaEnv->pDevMatchFilter, sizeof(Complex) * MATCHFILTER_BUFFER_SIZE));
	checkCudaErrors(hipMalloc((void**)&pCudaEnv->pDevRxData, sizeof(Complex) * DATA_BUFFER_SIZE));
	checkCudaErrors(hipMalloc((void**)&pCudaEnv->pDevPcData, sizeof(Complex) * DATA_BUFFER_SIZE));
	checkCudaErrors(hipMalloc((void**)&pCudaEnv->pDevMTDData, sizeof(Complex) * DATA_BUFFER_SIZE));
	checkCudaErrors(hipMalloc((void**)&pCudaEnv->pDevCfarData, sizeof(float) * DATA_BUFFER_SIZE));
	

	int total_size = 0;
	for (int win_size = 16; win_size <= 65536; win_size *= 2)
		total_size += win_size;

	checkCudaErrors(hipMalloc((void**)&pCudaEnv->pBlackmanWin, sizeof(float) * total_size));


	checkCudaErrors(hipStreamCreate(&pCudaEnv->stream0));
	checkCudaErrors(hipStreamCreate(&pCudaEnv->stream1));

	rsp_MatchFilter_Init(pCudaEnv);



}

void rsp_UnInit(RSP_CUDE_ENV* pCudaEnv)
{
	checkCudaErrors(hipHostFree(pCudaEnv->pHostMatchFilter));
	checkCudaErrors(hipHostFree(pCudaEnv->pHostRxData));
	checkCudaErrors(hipHostFree(pCudaEnv->pHostPcData));
	checkCudaErrors(hipHostFree(pCudaEnv->pHostMTDData));

	checkCudaErrors(hipFree(pCudaEnv->pDevMatchFilter));
	checkCudaErrors(hipFree(pCudaEnv->pDevRxData));
	checkCudaErrors(hipFree(pCudaEnv->pDevPcData));
	checkCudaErrors(hipFree(pCudaEnv->pDevMTDData));
	checkCudaErrors(hipFree(pCudaEnv->pDevCfarData));
	checkCudaErrors(hipFree(pCudaEnv->pBlackmanWin));

	checkCudaErrors(hipStreamDestroy(pCudaEnv->stream0));
	checkCudaErrors(hipStreamDestroy(pCudaEnv->stream0));


}


//
////��СCPIΪ8�����̴߳�(wrap)Ϊ32������̲߳���8x4�Ŀ�
////wrap�Ĵ�С����ͨ������wrapSize���
void rsp_MatchFilter(Complex* pDevBuf, Complex* pMatchFilter, int nRow, int nCol)
{
	//MatrixWiseMulVec(Complex* rx, const float *win, Complex* result, int row, int col, float scale)
	dim3 threadsPerBlock = {8,4};
	dim3 blocksPerGrid;

	blocksPerGrid.y = (nRow + 7) / 8;
	blocksPerGrid.x = (nCol + 3) / 4;
	MatrixWiseMulVecRow  <<< blocksPerGrid, threadsPerBlock >>> (pDevBuf, pMatchFilter, pDevBuf, nRow, nCol, 1.0f);

}
//
void rsp_PulseCompress(RSP_CUDE_ENV* pCudaEnv, Complex* pDevBuf, int nCode, int nRow, int nCol, hipStream_t stream)
{
	Complex *devMatchFilter;


	hipfftHandle plan_pc;
	size_t workSize;
	int new_size_long = nCol;
	int batch_size = nRow;

	int nembed[2] = { nRow , nCol };
	int stride = 1;
	int dist = nCol;


	for (int i = 0; i < pCudaEnv->MF_NUM; i++)
	{
		if (pCudaEnv->pMatchFilter[i].nCode == nCode)
		{
			UINT32 uMFOffset = pCudaEnv->pMatchFilter[i].uOffset;
			devMatchFilter = &pCudaEnv->pDevMatchFilter[uMFOffset];
		}
	}

	checkCudaErrors(hipfftCreate(&plan_pc));
	checkCudaErrors(hipfftMakePlanMany(plan_pc, 1, &new_size_long, nembed, stride, dist, nembed, stride, dist, HIPFFT_C2C, batch_size, &workSize));

	// Transform signal and kernel
	checkCudaErrors(hipfftExecC2C(plan_pc, (hipfftComplex *)pDevBuf, (hipfftComplex *)pDevBuf, HIPFFT_FORWARD));

	rsp_MatchFilter(pDevBuf, devMatchFilter, nRow, nCol);

	checkCudaErrors(hipfftExecC2C(plan_pc, (hipfftComplex *)pDevBuf, (hipfftComplex *)pDevBuf, HIPFFT_BACKWARD));


}

void rsp_Mtd(RSP_CUDE_ENV* pCudaEnv, Complex* pDevBuf, int nRow, int nCol, hipStream_t stream)
{
	
	dim3 threadsPerBlock = { 8,4 };
	dim3 blocksPerGrid;

	blocksPerGrid.y = (nRow + 7) / 8;
	blocksPerGrid.x = (nCol + 3) / 4;

	int total_size = 0;
	for (int win_size = 16; win_size <= nRow; win_size *= 2)
	{
		total_size += win_size;
	}

	float* pWin = &pCudaEnv->pBlackmanWin[total_size];

	MatrixWiseRealMulVecCol <<< blocksPerGrid, threadsPerBlock >>> (pDevBuf, pWin, pCudaEnv->pDevMTDData, nRow, nCol);

	hipfftHandle plan_adv;
	size_t workSize;
	int new_size_long = nRow;
	int batch_size = nCol;

	int nembed[2] = { nRow , nCol };
	int stride = nCol;
	int dist = 1;


	checkCudaErrors(hipfftCreate(&plan_adv));
	checkCudaErrors(hipfftMakePlanMany(plan_adv, 1, &new_size_long, nembed, stride, dist, nembed, stride, dist, HIPFFT_C2C, batch_size, &workSize));

	// Transform signal and kernel
	checkCudaErrors(hipfftExecC2C(plan_adv, (hipfftComplex *)pCudaEnv->pDevMTDData, (hipfftComplex *)pCudaEnv->pDevMTDData, HIPFFT_FORWARD));
	checkCudaErrors(hipfftDestroy(plan_adv));
	
}

//void __global__ rsp_Cfar(Complex* cfar_in, float* cfar_out, float* cfar_buf,  int nRow, int nCol, BOOL bAbsEn)
//{
//	int crow, ccol;
//	//int cstride;
//	int offset;
//	int cfar_stag;
//
//	int step[4];
//	int last[4];
//
//	crow = threadIdx.x;
//	ccol = blockIdx.x;
//	offset = crow*nCol + ccol;
//
//	crow = blockIdx.y * blockDim.y + threadIdx.y;
//	ccol = blockIdx.x * blockDim.x + threadIdx.x;
//	//cstride = blockDim.x * gridDim.x;
//	offset = crow * nCol + ccol;
//	cfar_stag = nRow * nCol;
//	step[0] = 1;
//	step[1] = 2;
//	step[2] = 4;
//	step[3] = 8;
//	last[0] = nCol - 1;
//	last[1] = nCol - 2;
//	last[2] = nCol - 4;
//	last[3] = nCol - 8;
//
//	if ((crow < nRow) && (ccol < nCol) && bAbsEn)
//	{
//		cfar_buf[offset] = ComplexAbs(cfar_in[offset]);
//		__syncthreads();
//	}
//	
//
//	if (ccol >= last[0])
//		cfar_buf[cfar_stag + offset] = 0.0f;
//	else
//		cfar_buf[cfar_stag + offset] = cfar_buf[offset] + cfar_buf[offset + step[0]];
//	__syncthreads();
//
//	cfar_stag += nRow * nCol;
//
//	if (ccol >= last[1])
//	cfar_buf[cfar_stag + offset] = 0.0f;
//	else
//		cfar_buf[cfar_stag + offset] = cfar_buf[offset] + cfar_buf[offset + step[1]];
//	__syncthreads();
//
//	cfar_stag += nRow * nCol;
//
//	if (ccol >= last[2])
//	cfar_buf[cfar_stag + offset] = 0.0f;
//	else
//		cfar_buf[cfar_stag + offset] = cfar_buf[offset] + cfar_buf[offset + step[2]];
//	__syncthreads();
//
//	cfar_stag += nRow * nCol;
//
//	if (ccol >= last[3])
//	cfar_buf[cfar_stag + offset] = 0.0f;
//	else
//		cfar_buf[cfar_stag + offset] = cfar_buf[offset] + cfar_buf[offset + step[3]];
//	__syncthreads();
//
//}

//
////�����״������鴦���������ڰ�CPI���鴦���GPU����ͨ���������ƴ�������
////��� pMatchFilterΪNULL����ʾ��������ѹ��
void rsp_pd_cpi(RSP_CUDE_ENV* pCudaEnv, void* pRadarRx, int nCode, int nRow, int nCol,  RSP_MTD_MODE rspMode)
{
		
	rsp_MemcpyToDevice(pCudaEnv->pDevRxData, pRadarRx,  nRow*nCol, pCudaEnv->stream0);
	

	rsp_PulseCompress(pCudaEnv, pCudaEnv->pDevRxData, nCode, nRow, nCol, pCudaEnv->stream0);

	rsp_Mtd(pCudaEnv, pCudaEnv->pDevRxData, nRow, nCol, pCudaEnv->stream0);

	switch (rspMode)
	{
		case RSP_MTD_MTI: break;
		case RSP_MTD_NCA: break;
		case RSP_MTD_MTD: break;
		case RSP_MTD_MTIMTD: break;
		case RSP_MTD_MTINCA:break;

		default:
			break;
	} 

	

}