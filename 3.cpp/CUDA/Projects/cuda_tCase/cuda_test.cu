#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>

#include "helper_functions.h"
#include <hip/hip_runtime_api.h>

#include "rsp_base.cuh"
#include "radar_process.h"

