#include "hip/hip_runtime.h"
/*
 * @Author: xpt 
 * @Date: 2019-12-12 11:28:26 
 * @Last Modified by: xpt
 * @Last Modified time: 2019-12-12 11:38:47
 */
#include<hip/hip_runtime.h>
#include<stdio.h>

void sumArrayOnHost(float *A, float *B, float *C, const int N)
{
    int idx = 0;
    for(; idx < N; idx++)
    {
        C[idx] = A[idx] + B[idx];
    }
}

__global__ void sumArrayOnGPU(float *A, float *B, float *C)
{
    int i = threadIdx.x;
    C[i] = A[i] + B[i];
}

void initialData(float *ip, int size)
{
    time_t t;
    srand((unsigned int) time(&t));
    for (int t = 0; t < size; t++)
    {
        ip[t] = (float)(rand() & 0xFF)/10.0f;
    }
}

void checkResult(float *hostRef, float *gpuRef, const int N)
{
    double epsilon = 1.0E-8;
    bool match = true;
    for(int i = 0; i < N; i++)
    {
        if(abs(hostRef[i] - gpuRef[i]) > epsilon)
        {
            match = false;
            printf("Arrays do not match!\n");
            break;
        }
    }
    if(match)
    {
        printf("Arrays  match\n\n");
    }
}

int main(int argc, char **argv)
{
    printf("%s Starting...\n", argv[0]);
    int dev = 0;
    hipSetDevice(dev);

    int nElem = 1024;
    int nBytes = nElem * sizeof(float);
    float *h_A, *h_B, *hostRef, *gpuRef;
    h_A = (float *)malloc(nBytes);
    h_B = (float *)malloc(nBytes);
    hostRef = (float *)malloc(nBytes);
    gpuRef = (float *)malloc(nBytes);
    initialData(h_A,nElem);
    initialData(h_B,nElem);
    memset(hostRef, 0 , nBytes);
    memset(gpuRef, 0 , nBytes);

    float *d_A, *d_B, *d_C;
    hipMalloc((float **)&d_A, nBytes);
    hipMalloc((float **)&d_B, nBytes);
    hipMalloc((float **)&d_C, nBytes);
    hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);

    dim3 block(nElem);
    dim3 grid((nElem + block.x - 1)/block.x);
    sumArrayOnGPU<<<grid, block>>>(d_A, d_B, d_C);
    printf("Execution configuration <<<%d, %d>>>\n", grid.x, block.x);
    hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);

    sumArrayOnHost(h_A, h_B, hostRef,nElem);

    checkResult(hostRef, gpuRef, nElem);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);
    return 0;
}