#include<hip/hip_runtime.h>
#include<stdio.h>
__global__ void CheckIndex(const int KernelNo)
{
    printf("KernelNo %d, threadIdx:(%d, %d, %d) blockIdx:(%d, %d, %d) blockDim:(%d, %d, %d) gridDim:(%d, %d, %d)\n",
            KernelNo,
            threadIdx.x, threadIdx.y, threadIdx.z,
            blockIdx.x, blockIdx.y, blockIdx.z,
            blockDim.x, blockDim.y, blockDim.z,
            gridDim.x, gridDim.y, gridDim.z);
}

int main(void)
{
    int nElm = 512;
    dim3 block(3);
    dim3 grid  ((nElm + block.x - 1)/block.x);
    printf("grid.x %d, grid.y %d, grid.z %d\n", grid.x, grid.y, grid.z);
    printf("block.x %d, block.y %d, block.z %d\n", block.x, block.y, block.z);
    CheckIndex<<<grid, block>>>(0);
    block.x = 2;
    grid.x = ((nElm + block.x - 1)/block.x);
    CheckIndex<<<grid, block>>>(1);
    hipDeviceReset();
    return 0;
}