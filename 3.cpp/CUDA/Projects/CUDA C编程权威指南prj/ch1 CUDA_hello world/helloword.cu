
#include <hip/hip_runtime.h>
#include<stdio.h>
__global__ void HelloFromGPU(void)
{
    printf("Hello World from GPU!\n");
}

int main(void)
{
    printf("Hello World from CPU!\n");
    HelloFromGPU<<<1,10>>>();
    hipDeviceReset();
    return 0;
}